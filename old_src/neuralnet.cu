#include "hip/hip_runtime.h"



/// Matrix multiplication for a layer.
__global__ void layer(float* input, float* matrix, float* output, int input_size) {
    int i = threadIdx.x;
    output[i / input_size] += input[i % input_size] * matrix[i];
}

/// Bias and normalization for a hidden layer.
__global__ void hidden_normalization(float* output, float* bias) {
    int i = threadIdx.x;
    output[i] = tanhf(output[i] + bias[i]);
}

/// Bias and normalization for output layer.
__global__ void output_normalization(float* output, float* bias) {
    int i = threadIdx.x;
    output[i] = (tanhf(output[i] + bias[i]) + 1) / 2; //sigmoid
}


extern "C" {
    /// Runs feed-forward propogation on the given input.
    void feed_forward(float* board, //42 nodes
            float* layer0, float* bias0, //84 nodes
            float* layer1, float* bias1, //21 nodes
            float* layer2, float* bias2, //7 nodes
            float* output) {
        // board (42) -> (84) -> (21) -> output (7)
        // board[42], layer0[3528], bias0[84], layer1[1764], bias1[21], layer2[147], bias2[7], output[7]
        
        float* cudaVec; //holds the left operands
        float* cudaMat; //holds the right operands
        float* cudaOut; //holds the intermediate outputs
        float* cudaBias; //holds the biases

        // move inputs into memory
        hipMalloc(&cudaVec, 42 * sizeof(float));
        hipMalloc(&cudaMat, 3528 * sizeof(float));
        hipMalloc(&cudaOut, 84 * sizeof(float));
        hipMalloc(&cudaBias, 84 * sizeof(float));
        hipMemcpy(cudaVec, board, 42 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(cudaMat, layer0, 3528 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(cudaBias, bias0, 84 * sizeof(float), hipMemcpyHostToDevice);

        // first layer
        layer<<<1, 3528>>>(cudaVec, cudaMat, cudaOut, 42); 
        hidden_normalization<<<1, 84>>>(cudaOut, cudaBias);

        // deallocation
        hipFree(cudaVec);
        hipFree(cudaMat);
        hipFree(cudaBias);

        // prepare inputs for second layer
        cudaVec = cudaOut;
        hipMalloc(&cudaMat, 1764 * sizeof(float));
        hipMalloc(&cudaOut, 21 * sizeof(float));
        hipMalloc(&cudaBias, 21 * sizeof(float));
        hipMemcpy(cudaMat, layer1, 1764 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(cudaBias, bias1, 21 * sizeof(float), hipMemcpyHostToDevice);

        // second layer
        layer<<<1, 1764>>>(cudaVec, cudaMat, cudaOut, 84); 
        hidden_normalization<<<1, 21>>>(cudaOut, cudaBias);

        // deallocation
        hipFree(cudaVec);
        hipFree(cudaMat);
        hipFree(cudaBias);

        // prepare inputs for third layer
        cudaVec = cudaOut;
        hipMalloc(&cudaMat, 147 * sizeof(float));
        hipMalloc(&cudaOut, 7 * sizeof(float));
        hipMalloc(&cudaBias, 7 * sizeof(float));
        hipMemcpy(cudaMat, layer2, 147 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(cudaBias, bias2, 7 * sizeof(float), hipMemcpyHostToDevice);

        // third layer
        layer<<<1, 147>>>(cudaVec, cudaMat, cudaOut, 21); 
        output_normalization<<<1, 7>>>(cudaOut, cudaBias);

        // retrieve and deallocate
        hipMemcpy(output, cudaOut, 7 * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(cudaVec);
        hipFree(cudaMat);
        hipFree(cudaOut);
        hipFree(cudaBias);
    }
}

